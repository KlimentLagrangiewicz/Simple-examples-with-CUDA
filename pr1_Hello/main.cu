#include <stdio.h>

#include "hip/hip_runtime.h"


__global__ void cuda_hello() {
    printf("Hello World from GPU!\n");
}

/*
    compiling:
        nvcc main.cu -o main
    launching:
        ./main
*/

int main(int argc, char **argv) {
    printf("Hello World from CPU!\n");
    cuda_hello<<<1,10>>>();
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}


